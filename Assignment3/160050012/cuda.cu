
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using std::cout;
using namespace std;

__global__ void matrixMul(float *a, float *b, float *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < N && col < N){
    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
      c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
  }
}

void init_matrix(float *m, int N){
  for (int i=0;i<N*N;i++){
    m[i] = ((float) rand()) / (float) RAND_MAX ;
  }
}

void print_matrix(float * a, int N){
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cout << a[i*N+j]<<"\t";
    }
    cout<<"\n";
  }
}


int main(int argc, char* argv[]) {

  std::cout << std::fixed;
  std::cout << std::setprecision(4);
  int N = atoi(argv[1]);
 
  // host memory
  float *host_A, *host_B , *host_C;
  host_A = new float[N*N];
  host_B = new float[N*N];
  host_C = new float[N*N];
  
  // Threads per CTA dimension
  int THREADS = 32;
  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = (N+THREADS-1)/THREADS;
  
  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Initialize matrices
  init_matrix(host_A,N);
  init_matrix(host_B,N);

  // Allocate device memory
  float *device_A, *device_B , *device_C;
  auto s = chrono::steady_clock::now();
  hipMalloc(&device_A, N*N*sizeof(float));
  hipMalloc(&device_B, N*N*sizeof(float));
  hipMalloc(&device_C, N*N*sizeof(float));

  // Copy data to the device
  hipMemcpy(device_A, host_A, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, N*N*sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(device_A, device_B, device_C,N);
  
  // Copy back to the host
  hipMemcpy(host_C, device_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
 
  // Free memory on device
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
  
  // Note Time
  auto e = chrono::steady_clock::now();
  auto diff = e - s;
  double mSecs =chrono::duration <double, milli> (diff).count();

  // Printing
  cout <<"Size = "<<N<<"\t";
  cout<<"\n \n";
  cout<<"A = \n";
  print_matrix(host_A,N);
  cout<<"\n";
  cout<<"B = \n";
  print_matrix(host_B,N);
  cout<<"\n";
  cout<<"C = \n";
  print_matrix(host_C,N);
  cout<<"\n";
  cout<<"Time in ms = "<<mSecs;
  cout<<"\n";
  return 0;
}