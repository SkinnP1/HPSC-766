#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using std::cout;
using namespace std;

__global__ void matrixMul(float *a, float *b, float *c, int N) {
  printf("%s","Hello World");
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < N && col < N){
    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
      c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
    printf("%f",c[row * N + col]);
  }
}

void init_matrix(float *m, int N){
  for (int i=0;i<N*N;i++){
    m[i] = ((float) rand()) / (float) RAND_MAX ;
  }
}

void print_matrix(float * a, int N){
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cout << a[i*N+j]<<"\t";
    }
    cout<<"\n";
  }
}

void verify_result(float *a, float *b, float *c, int N){
    float tmp;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            tmp = 0;
            for(int k = 0; k < N; k++){
                tmp += a[i * N + k] * b[k * N + j];
            }
            assert(tmp == c[i * N + j]);
        }
    }
}

int main(int argc, char* argv[]) {

  int N = atoi(argv[1]);
  std::cout << std::fixed;
  std::cout << std::setprecision(5);
  // host memory
  float *host_A, *host_B , *host_C;
  host_A = new float[N*N];
  host_B = new float[N*N];
  host_C = new float[N*N];

  // Initialize matrices
  init_matrix(host_A,N);
  init_matrix(host_B,N);

  // Allocate device memory
  float *device_A, *device_B , *device_C;
  hipMalloc(&device_A, N*N*sizeof(float));
  hipMalloc(&device_B, N*N*sizeof(float));
  hipMalloc(&device_C, N*N*sizeof(float));

  // Copy data to the device
  hipMemcpy(device_A, host_A, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, N*N*sizeof(float), hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 16;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = (N+THREADS-1)/THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  auto s = chrono::steady_clock::now();
  matrixMul<<<blocks, threads>>>(device_A, device_B, device_C,N);
  auto e = chrono::steady_clock::now();
  auto diff = e - s;
  double mSecs =chrono::duration <double, milli> (diff).count();

  // Copy back to the host
  hipMemcpy(host_C, device_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

  cout << "COMPLETED SUCCESSFULLY\n";
  verify_result(host_A, host_B, host_C, N);

  // Free memory on device
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);

  cout <<"Size = "<<N;
  cout<<"\n \n";
  cout<<"A = \n";
  print_matrix(host_A,N);
  cout<<"\n";
  cout<<"B = \n";
  print_matrix(host_B,N);
  cout<<"\n";
  cout<<"C = \n";
  print_matrix(host_C,N);
  cout<<"\n";
  cout<<"Time in ms = "<<mSecs;
  cout<<"\n";
  return 0;
}