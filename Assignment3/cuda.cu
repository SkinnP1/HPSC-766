
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define N 10
using std::cout;

__global__ void matrixMul(float *a, float *b, float *c) {
  printf("%s","Hello World");
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < N && col < N){
    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
      c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
    printf("%f",c[row * N + col]);
  }
}

void init_matrix(float *m){
  for (int i=0;i<N*N;i++){
    m[i] = ((float) rand()) / (float) RAND_MAX ;
  }
}

void print_matrix(float * a){
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cout << a[i*N+j]<<"\t";
    }
    cout<<"\n";
  }
}

int main() {

  // host memory
  float *host_A, *host_B , *host_C;
  host_A = new float[N*N];
  host_B = new float[N*N];
  host_C = new float[N*N];

  // Initialize matrices
  init_matrix(host_A);
  init_matrix(host_B);

  // Allocate device memory
  float *device_A, *device_B , *device_C;
  hipMalloc(&device_A, N*N*sizeof(float));
  hipMalloc(&device_B, N*N*sizeof(float));
  hipMalloc(&device_C, N*N*sizeof(float));

  // Copy data to the device
  hipMemcpy(device_A, host_A, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, N*N*sizeof(float), hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 16;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = (N+THREADS-1)/THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(device_A, device_B, device_C);

  // Copy back to the host
  hipMemcpy(host_C, device_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

  cout << "COMPLETED SUCCESSFULLY\n";



  // Free memory on device
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);

  // print_matrix(host_A);
  // print_matrix(host_B);
  // print_matrix(host_C);
  return 0;
}